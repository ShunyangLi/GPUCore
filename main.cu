/**
 * @author Shunyang Li
 * Contact: sli@cse.unsw.edu.au
 * @date on 2024/6/12.
 * @brief: main function
 */
#include <argparse/argparse.hpp>
#include <iostream>

#include "util/utility.h"
#include "graph/graph.h"
#include "core/core.cuh"

int main(int argc, char* argv[]) {

    argparse::ArgumentParser parser("core", "1.0.0");
    add_args(parser);

    std::locale loc("");
    std::locale::global(loc);

    try {
        parser.parse_args(argc, argv);
    } catch (const std::exception& err) {
        log_error("error: %s", err.what());
        std::cout << parser << std::endl;
        exit(EXIT_FAILURE);
    }

    auto device_count = 0;
    auto device_id = 0;

    hipGetDeviceCount(&device_count);
    if (device_count == 0) log_warn("no gpu devices supporting CUDA.");

    if (parser.is_used("--device")) {
        device_id = parser.get<int>("--device");
        if (device_id >= device_count) {
            log_error("error: gpu device id %d is not available", device_id);
            exit(EXIT_FAILURE);
        }
        hipSetDevice(device_id);
    }

    if (parser.get<bool>("--device_info")) {
        if (device_count == 0) log_warn("no gpu devices supporting CUDA.");
        else
            get_device_info(device_id);
    }

    if (parser.is_used("--graph")) {

        // convert the graph file to binary file
        if (parser.is_used("--bin")) {
            const std::string& filename = parser.get<std::string>("--bin");
            const std::string& dataset = parser.get<std::string>("--graph");

            auto g = Graph(dataset, true);
            g.graph_to_bin(filename);
            return 0;
        }


        auto dataset = parser.get<std::string>("--graph");
        auto g = Graph(dataset, false);

        auto pair = std::vector<std::pair<int, int>>({
                {1, 1},
                {2, 1},
                {2, 2},
                {3, 5},
                {10, 30},
                {55, 12},
                {4, 22},
                {20, 11}
        });

        for (auto& p : pair) {
            auto alpha = p.first;
            auto beta = p.second;
            g_abcore_peeling(&g, alpha, beta);
            c_abcore_peeling(g, alpha, beta);
        }

    }


    return 0;
}
