#include "hip/hip_runtime.h"
/**
 * @author Shunyang Li
 * Contact: sli@cse.unsw.edu.au
 * @date on 2024/10/10.
 */


#include "core.cuh"

__global__ auto scan_kernel(const int* d_degree, uint* buf_tails, uint* g_buffers,
                            int alpha, int beta, uint u_num, uint num_vertex) -> void {

    __shared__ uint* g_buffer;
    __shared__ uint bufTail;

    if (threadIdx.x == 0) {
        bufTail = 0;
        g_buffer = g_buffers + blockIdx.x * GLBUFFER_SIZE;
    }
    __syncthreads();

    uint g_idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (uint base = 0; base < num_vertex; base += N_THREADS) {
        uint v = base + g_idx;
        if (v >= num_vertex) continue;

        uint threshold = v < u_num ? alpha : beta;

        if (d_degree[v] < threshold) {
            uint idx = atomicAdd(&bufTail, 1);
            writeToBuffer(g_buffer, idx, v);
        }
    }

    __syncthreads();

    if (threadIdx.x == 0) {
        buf_tails[blockIdx.x] = bufTail;
    }
}


__global__ auto peel_kernel(const uint* d_offset, const uint* d_neighbors, int* d_degree,
                            const uint* buf_tails, uint* g_buffers,
                            uint u_num, int alpha, int beta) -> void {

    __shared__ uint buf_tail;
    __shared__ uint* g_buffer;
    __shared__ uint base;

    uint warp_id = threadIdx.x / 32;
    uint lane_id = threadIdx.x % 32;
    uint regTail;
    uint i;

    if (threadIdx.x == 0) {
        buf_tail = buf_tails[blockIdx.x];
        base = 0;
        g_buffer = g_buffers + blockIdx.x * GLBUFFER_SIZE;
    }


    while (true) {
        __syncthreads();
        // all the threads will evaluate to true at same iteration
        if (base == buf_tail) break;
        i = base + warp_id;
        regTail = buf_tail;

        __syncthreads();

        if (i >= regTail) continue;// this warp won't have to do anything

        if (threadIdx.x == 0) {
            // update base for next iteration
            base += WARPS_EACH_BLK;
            if (regTail < base) base = regTail;
        }

        //bufTail is incremented in the code below:
        uint v = readFromBuffer(g_buffer, i);

        uint start = d_offset[v];
        uint end = d_offset[v + 1];

        while (true) {
            __syncwarp();

            if (start >= end) break;

            uint j = start + lane_id;
            start += WARP_SIZE;
            if (j >= end) continue;

            uint u = d_neighbors[j];
            int threshold = u < u_num ? alpha : beta;

            int deg_u = atomicSub(d_degree + u, 1);

            if ((deg_u - 1) == (threshold - 1)) {
                uint loc = atomicAdd(&buf_tail, 1);
                writeToBuffer(g_buffer, loc, u);
            }
        }
    }
}

/**
 * abcore online peeling algorithm on gpu
 * @param g graph
 * @param alpha alpha value
 * @param beta beta value
 */
auto g_abcore_peeling(Graph* g, int alpha, int beta) -> void {

    log_info("running (alpha,beta)-core online peeling algorithm on GPU");

    auto left_degree_max = std::max_element(g->degrees, g->degrees + g->u_num);
    auto right_degree_max = std::max_element(g->degrees + g->u_num, g->degrees + g->n - 1);

    // check if the graph is valid
    if (*left_degree_max < alpha || *right_degree_max < beta) {
        log_error("max degree: (%d, %d), query (%d, %d) is not valid", *left_degree_max, *right_degree_max, alpha, beta);
        return;
    }

    uint* d_offset;
    uint* d_neighbors;
    int* d_degree;
    uint* g_buffers;
    uint* buf_tails;

    CER(hipMalloc(&d_offset, sizeof(uint) * (g->n + 1)));
    CER(hipMalloc(&d_neighbors, sizeof(uint) * g->m * 2));
    CER(hipMalloc(&d_degree, sizeof(int) * g->n));
    CER(hipMalloc(&buf_tails, sizeof(uint) * BLK_NUMS));
    CER(hipMalloc(&g_buffers, sizeof(uint) * GLBUFFER_SIZE * BLK_NUMS));


    CER(hipMemcpy((void*) d_offset, (void*) g->offsets, sizeof(uint) * (g->n + 1), hipMemcpyHostToDevice));
    CER(hipMemcpy((void*) d_neighbors, (void*) g->neighbors, sizeof(uint) * g->m * 2, hipMemcpyHostToDevice));
    CER(hipMemcpy((void*) d_degree, (void*) g->degrees, sizeof(uint) * g->n, hipMemcpyHostToDevice));
    CER(hipMemset((void*) buf_tails, 0, sizeof(uint) * BLK_NUMS));


    auto timer = new Timer();
    timer->reset();

    scan_kernel<<<BLK_NUMS, BLK_DIM>>>(d_degree, buf_tails, g_buffers, alpha, beta, g->u_num, g->n);
    peel_kernel<<<BLK_NUMS, BLK_DIM>>>(d_offset, d_neighbors, d_degree, buf_tails, g_buffers, g->u_num, alpha, beta);

    hipDeviceSynchronize();

    auto time = timer->elapsed();
    log_info("abcore peeling time on gpu: %f s", time);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        log_trace("CUDA error: %s", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    auto upper_vertices = std::vector<uint>();
    auto lower_vertices = std::vector<uint>();
    auto degrees = new int[g->n];

    // get degrees
    hipMemcpy((void*) degrees, (void*) d_degree, sizeof(int) * g->n, hipMemcpyDeviceToHost);

    // get result
    for (auto i = 0; i < g->u_num; i++)
        if (degrees[i] >= alpha) upper_vertices.push_back(i);
    for (auto i = g->u_num; i < g->n; i++)
        if (degrees[i] >= beta) lower_vertices.push_back(i);

    // free cuda memory
    hipFree(d_offset);
    hipFree(d_neighbors);
    hipFree(d_degree);
    hipFree(buf_tails);
    hipFree(g_buffers);

    delete timer;

#ifdef DISPLAY_RESULT
    log_info("upper vertices: %d, lower vertices: %d", upper_vertices.size(), lower_vertices.size());
#endif
}